#include "hip/hip_runtime.h"
#include <mex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess) {                                                              \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        }                                                                                    \
    }

__global__ void hipLaunchByPtr(float* imData, float* output, int nSize, int searchSize, mwSize dx, mwSize dy, mwSize dz)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y,
        z = threadIdx.z + blockIdx.z * blockDim.z,
        coord[3] = { x + nSize, y + nSize, z + nSize },
        imDims[3] = { dx, dy, dz };

    if (x >= (imDims[0] - 2 * nSize) || y >= (imDims[1] - 2 * nSize) || z >= (imDims[2] - 2 * nSize) || 
        (int)imData[coord[2] * imDims[0] * imDims[1] + coord[1] * imDims[0] + coord[0]] == 0) {
        
        return;
    }

    int minc[3], maxc[3];
    long int offset = 25 * (x + y * (dx - 2 * nSize) + z * (dx - 2 * nSize) * (dy - 2 * nSize));
    nSize >>= 1;

    int i;
    for (i = 0; i < 3; ++i) {
        minc[i] = coord[i] + 1 - searchSize / 2;
        maxc[i] = minc[i] + searchSize;
        if (minc[i] < nSize)
            minc[i] = nSize;
        
        if (maxc[i] > imDims[i] - nSize)
            maxc[i] = imDims[i] - nSize;
    }

    float diff, val, dist[25];
    // Find 25 closest neighbors. Initialize initial distances to INF.
    for (i = 0; i < 25; i++)
        dist[i] = 1E+37; 

    for (z = minc[2]; z < maxc[2]; ++z) {
        for (y = minc[1]; y < maxc[1]; ++y) {
            for (x = minc[0]; x < maxc[0]; ++x) {
                if (x == coord[0] && y == coord[1] && z == coord[2])
                    continue;

                long int bi = z * imDims[0] * imDims[1] + y * imDims[0] + x;
                long int bj = coord[2] * imDims[0] * imDims[1] + coord[1] * imDims[0] + coord[0];

                diff = 0;
                long int rel1 = -nSize * imDims[0] * imDims[1] - nSize * imDims[0] - nSize;
                for (dz = -nSize; dz <= nSize; ++dz) {
                    long int rel2 = rel1;
                    for (dy = -nSize; dy <= nSize; ++dy) {
                        long int rel3 = rel2;
                        for (dx = -nSize; dx <= nSize; ++dx) {
                            val = imData[bi + rel3] - imData[bj + rel3];
                            if (dz || dy || dx)
                                diff += (val * val);
                            rel3++;
                        }
                        rel2 += imDims[0];
                    }
                    rel1 += imDims[0] * imDims[1];
                }

                // Minor fix for the crash here.
                for (ii = 24; ii > 0 && diff < dist[ii]; ii--) {
                    dist[ii] = dist[ii - 1];
                    output[offset + ii] = output[offset + ii - 1];
                }

                if (ii == 0) {
                    dist[0] = diff;
                    output[offset] = imData[bi];
                }
                else if (ii != 24) {
                    dist[ii + 1] = diff;
                    output[offset + ii + 1] = imData[bi];
                }
            }
        }
    }
}

void mexFunction(int nlhs, mxArray* plhs[],
    int nrhs, const mxArray* prhs[])
{
    const mwSize* imDims = mxGetDimensions(prhs[0]);
    float* imData = (float*)mxGetData(prhs[0]);
    double* searchSize = mxGetPr(prhs[1]);
    double* neighSize = mxGetPr(prhs[2]);

    int dims[2] = { 25 * (imDims[2] - 2 * neighSize[2]) * (imDims[1] - 2 * neighSize[1]) * (imDims[0] - 2 * neighSize[0]), 1 };
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float* output = (float*)mxGetData(plhs[0]);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *d_imData, *d_output, executionTime;

    hipEventRecord(start, 0);
    hipMalloc((void**)&d_output, dims[0] * sizeof(float));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);
    mexPrintf("\n1. Execution time: gpu mem allocation for output gpuArray %f", executionTime);

    //	EXECUTION TIME-> MEMORY ALLOCATION ON GPU + TRANSFER OF DATA FROM RAM TO GPU MEMORY.
    hipEventRecord(start, 0);
    hipMalloc((void**)&d_imData, imDims[0] * imDims[1] * imDims[2] * sizeof(float));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);
    mexPrintf("\n2. Execution time: gpu mem allocation for imageData on GPU memory %f", executionTime);

    hipEventRecord(start, 0);
    hipMemcpy(d_imData, imData, imDims[0] * imDims[1] * imDims[2] * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);
    mexPrintf("\n3. Execution time: ImageData transfer from RAM to GPU memory %f", executionTime);
    // EXECUTION TIME-> MEMORY ALLOCATION ON GPU + TRANSFER OF DATA FROM RAM TO GPU MEMORY.

    dim3 grid((imDims[0] - 2 * neighSize[0] + 7) / 8, (imDims[1] - 2 * neighSize[1] + 7) / 8, (imDims[2] - 2 * neighSize[2] + 3) / 4),
        block(8, 8, 4);
    hipLaunchByPtr<<<grid, block> > >(d_imData, d_output, (int)neighSize[0], (int)searchSize[0], imDims[0], imDims[1], imDims[2]);
    hipDeviceSynchronize();

    //	EXECUTION TIME-> COMPUTED RESULTS FROM GPU MEMORY TO RAM
    hipEventRecord(start, 0);
    hipMemcpy(output, d_output, dims[0] * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);
    mexPrintf("\n4. Execution time: gpuArray transfer from GPU memory to RAM %f\n", executionTime);
    //	EXECUTION TIME-> COMPUTED RESULTS FROM GPU MEMORY TO RAM

    cudaCheckError();
    hipFree(d_output);
    hipFree(d_imData);
}
